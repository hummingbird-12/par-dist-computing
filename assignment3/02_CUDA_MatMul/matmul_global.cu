#include "hip/hip_runtime.h"
#include <stdio.h>

#include "matmul.cuh"

__global__ static void kernel(float* mat_a, float* mat_b, float* mat_c);

__host__ void matmul_global(const int BLOCK_SIZE, const float mat_a[][MAT_SIZE],
                            const float mat_b[][MAT_SIZE],
                            float mat_c[][MAT_SIZE]) {
    const int SIZE = sizeof(float) * MAT_SIZE * MAT_SIZE;
    float *dev_a, *dev_b, *dev_c;
    double start, end;

    const int GRID_DIM = MAT_SIZE / BLOCK_SIZE;
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(GRID_DIM, GRID_DIM);

    GET_TIME(start);

    hipMalloc((void**) &dev_a, SIZE);
    hipMalloc((void**) &dev_b, SIZE);
    hipMalloc((void**) &dev_c, SIZE);

    hipMemcpy(dev_a, mat_a, SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, mat_b, SIZE, hipMemcpyHostToDevice);
    hipMemset(dev_c, 0, SIZE);

    kernel<<<grid, block>>>(dev_a, dev_b, dev_c);
    hipDeviceSynchronize();

    hipMemcpy(mat_c, dev_c, SIZE, hipMemcpyDeviceToHost);

    GET_TIME(end);

    printf("[matmul_global]\tTime: %fs\n", end - start);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}

__global__ static void kernel(float* mat_a, float* mat_b, float* mat_c) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = 0; i < MAT_SIZE; i++) {
        mat_c[row * MAT_SIZE + col] +=
            mat_a[row * MAT_SIZE + i] * mat_b[i * MAT_SIZE + col];
    }
}
