#include "hip/hip_runtime.h"
#include <stdio.h>

#include "matmul.cuh"

__global__ static void kernel(float* mat_a, float* mat_b, float* mat_c);

__host__ void matmul_shared(const int BLOCK_DIM, const float mat_a[][MAT_SIZE],
                            const float mat_b[][MAT_SIZE],
                            float mat_c[][MAT_SIZE]) {
    const int SIZE = sizeof(float) * MAT_SIZE * MAT_SIZE;
    float *dev_a, *dev_b, *dev_c;
    double start, end;

    const int GRID_DIM = MAT_SIZE / BLOCK_DIM;
    dim3 block(BLOCK_DIM, BLOCK_DIM);
    dim3 grid(GRID_DIM, GRID_DIM);

    GET_TIME(start);

    hipMalloc((void**) &dev_a, SIZE);
    hipMalloc((void**) &dev_b, SIZE);
    hipMalloc((void**) &dev_c, SIZE);

    hipMemcpy(dev_a, mat_a, SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, mat_b, SIZE, hipMemcpyHostToDevice);
    hipMemset(dev_c, 0, SIZE);

    kernel<<<grid, block,
             sizeof(float) * BLOCK_DIM * BLOCK_DIM*(2 * GRID_DIM + 1)>>>(
        dev_a, dev_b, dev_c);
    hipDeviceSynchronize();

    hipMemcpy(mat_c, dev_c, SIZE, hipMemcpyDeviceToHost);

    GET_TIME(end);

    printf("[matmul_shared] Time: %fs\n", end - start);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}

__global__ static void kernel(float* mat_a, float* mat_b, float* mat_c) {
    extern __shared__ float shared[];
    float* shared_a = shared;  // [blockDim.y][MAT_SIZE]
    float* shared_b =
        shared_a + blockDim.y * MAT_SIZE;  // [MAT_SIZE][blockDim.x]
    float* shared_c =
        shared_b + MAT_SIZE * blockDim.x;  // [blockDim.y][blockDim.x]

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x * blockDim.x + threadIdx.y;

    shared_c[tid] = 0.0f;
    for (int i = 0; i < MAT_SIZE; i++) {
        shared_a[threadIdx.y * MAT_SIZE + i] = mat_a[row * MAT_SIZE + i];
        shared_b[i * blockDim.x + threadIdx.x] = mat_b[i * MAT_SIZE + col];
        shared_c[tid] += shared_a[threadIdx.y * MAT_SIZE + i] *
                         shared_b[i * blockDim.x + threadIdx.x];
    }
    mat_c[row * MAT_SIZE + col] = shared_c[tid];
}
