#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#include "max_reduction.cuh"

__global__ static void kernel(int arr[ARR_SIZE], int mx);

__host__ void reduction_divergent(const int arr[ARR_SIZE]) {
    const int SIZE = sizeof(int) * ARR_SIZE;
    int *dev_arr, *dev_mx, mx;
    double start, end;

    const int GRID_DIM = ceil((float) ARR_SIZE / DEF_BLOCK_SIZE);
    dim3 block(DEF_BLOCK_SIZE);
    dim3 grid(GRID_DIM);

    GET_TIME(start);

    hipMalloc((void**) &dev_arr, SIZE);
    hipMalloc((void**) &dev_mx, sizeof(int));

    hipMemcpy(dev_arr, arr, SIZE, hipMemcpyHostToDevice);
    hipMemset(dev_mx, 0, sizeof(int));

    kernel<<<grid, block>>>(dev_arr, dev_mx);
    hipDeviceSynchronize();

    hipMemcpy(&mx, dev_mx, sizeof(int), hipMemcpyDeviceToHost);

    GET_TIME(end);

    printf("[reduction_divergent] Maximum: %d\tTime: %fs\n", mx, end - start);

    hipFree(dev_arr);
    hipFree(dev_mx);
}

__global__ static void kernel(int arr[ARR_SIZE], int mx) {
    while (n > 1) {
	    int stride = (n + 1) / 2; // round up to find memory offset
	    int srcIdx = i + stride;
	    if (srcIdx < n) {
		    arr[i] += arr[srcIdx];
        }
	    barrier(CLK_GLOBAL_MEM_FENCE); /* subtle: needed so we can read newly added values */
	    n = stride; /* new size is everything except what we've already read */
   }
}
