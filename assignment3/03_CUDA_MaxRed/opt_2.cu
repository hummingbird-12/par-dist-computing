#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#include "max_reduction.cuh"

__global__ static void kernel(int* arr, int* n, int* stride);

__host__ void reduction_opt_2(const int* arr, const int n, const int BLOCK_SIZE) {
    const int SIZE = sizeof(int) * n;
    int mx;
    int *dev_arr, *dev_n, *dev_stride;
    double start, end;

    const int GRID_DIM = ceil((float) n / BLOCK_SIZE);
    dim3 block(BLOCK_SIZE);
    dim3 grid(GRID_DIM);

    GET_TIME(start);

    hipMalloc((void**) &dev_arr, SIZE);
    hipMalloc((void**) &dev_n, sizeof(int));
    hipMalloc((void**) &dev_stride, sizeof(int));

    hipMemcpy(dev_arr, arr, SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dev_n, &n, sizeof(int), hipMemcpyHostToDevice);

    for (int i = n, stride = (n + 1) / 2;
         i >= 1;
         i /= 2, stride = (stride + 1) / 2) {
        hipMemcpy(dev_stride, &stride, sizeof(int), hipMemcpyHostToDevice);
        kernel<<<grid, block, SIZE>>>(dev_arr, dev_n, dev_stride);
        hipDeviceSynchronize();
    }

    hipMemcpy(&mx, dev_arr, sizeof(int), hipMemcpyDeviceToHost);

    GET_TIME(end);

    printf("[reduction_opt_2]\tMaximum: %d\tTime: %fs\n", mx, end - start);

    hipFree(dev_arr);
    hipFree(dev_n);
    hipFree(dev_stride);
}

__global__ static void kernel(int* arr, int* n, int* stride) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ int shared[];

    if (tid < *stride && tid + *stride < *n) {
        shared[tid] = arr[tid];
        shared[tid + *stride] = arr[tid + *stride];
        arr[tid] = max(shared[tid], shared[tid + *stride]);
    }
}
